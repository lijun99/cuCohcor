#include "hip/hip_runtime.h"
#include "cuAmpcorChunk.h"
#include "cuAmpcorUtil.h"

/**
 * Run ampcor process for a batch of images (a chunk)
 * @param[in] idxDown_  index of the chunk along Down/Azimuth direction
 * @param[in] idxAcross_ index of the chunk along Across/Range direction
 */
void cuAmpcorChunk::run(int idxDown_, int idxAcross_)
{
    // set chunk index
    setIndex(idxDown_, idxAcross_);

    // load reference image chunk
    loadReferenceChunk();

#ifdef CUAMPCOR_DEBUG
    // dump the raw reference image(s)
    c_referenceBatchRaw->outputToFile("c_referenceBatchRaw", stream);
#endif

    // deramp ; 1=linear, others = none
    cuDeramp(param->derampMethod, c_referenceBatchRaw, stream);

#ifdef CUAMPCOR_DEBUG
    // dump the raw reference image(s)
    c_referenceBatchRaw->outputToFile("c_referenceBatchedRawDeramped", stream);
#endif

    // compute and subtract mean values (for normalized)
    cuArraysSubtractMean(c_referenceBatchRaw, stream);

#ifdef CUAMPCOR_DEBUG
    // dump the raw reference image(s)
    c_referenceBatchRaw->outputToFile("c_referenceBatchRawSubMean", stream);
#endif

    // whether to apply window filter on reference
    if(param->useHannWindowReference)
    {
        // make a copy of the reference images since they will be used again
        cuArraysCopy(c_referenceBatchRaw, c_referenceBatchRawFiltered, stream);
        windowFilterReferenceRaw->filter(c_referenceBatchRawFiltered, stream);

#ifdef CUAMPCOR_DEBUG
        // dump the filtered raw reference image(s)
        c_referenceBatchRawFiltered->outputToFile("c_referenceBatchRawFiltered", stream);
#endif
    }

    // load secondary image chunk
    loadSecondaryChunk();

#ifdef CUAMPCOR_DEBUG
    // dump the raw secondary image(s)
    c_secondaryBatchRaw->outputToFile("c_secondaryBatchRaw", stream);
#endif

    // deramp ; 1=linear, others = none
    cuDeramp(param->derampMethod, c_secondaryBatchRaw, stream);

#ifdef CUAMPCOR_DEBUG
    // dump the raw reference image(s)
    c_secondaryBatchRaw->outputToFile("c_secondaryBatchRawSubMean", stream);
#endif

    // whether to apply window filter on secondary
    if(param->useHannWindowSecondary)
    {
        // make a copy of the reference images since they will be used again
        cuArraysCopy(c_secondaryBatchRaw, c_secondaryBatchRawFiltered, stream);
        windowFilterSecondaryRaw->filter(c_secondaryBatchRawFiltered, stream);

#ifdef CUAMPCOR_DEBUG
        // dump the filtered raw secondary image(s)
        c_secondaryBatchRawFiltered->outputToFile("c_secondaryBatchRawFiltered", stream);
#endif
    }

    //cross correlation for un-oversampled data
    cuCorrFreqDomain->execute(c_referenceBatchRawFiltered, c_secondaryBatchRawFiltered, r_corrBatchRaw);

#ifdef CUAMPCOR_DEBUG
    // dump the un-normalized correlation surface
    r_corrBatchRaw->outputToFile("r_corrBatchRawUnNorm", stream);
#endif

    // normalize the correlation surface
    corrNormalizerRaw->execute(r_corrBatchRaw, c_referenceBatchRawFiltered, c_secondaryBatchRawFiltered, stream);

#ifdef CUAMPCOR_DEBUG
    // dump the normalized correlation surface
    r_corrBatchRaw->outputToFile("r_corrBatchRaw", stream);
#endif

    // find the maximum location of none-oversampled correlation
    // 41 x 41, if halfsearchrange=20
    cuArraysMaxloc2D(r_corrBatchRaw, offsetInit, r_maxval, stream);

    // estimate variance
    cuEstimateVariance(r_corrBatchRaw, offsetInit, r_maxval, c_referenceBatchRaw->size, r_covValue, stream);

    // estimate SNR
    // step1: extraction of correlation surface around the peak
    cuArraysCopyExtractCorr(r_corrBatchRaw, r_corrBatchRawZoomIn, i_corrBatchZoomInValid, offsetInit, stream);

    // step2: summation of correlation and data point values
    cuArraysSumCorr(r_corrBatchRawZoomIn, i_corrBatchZoomInValid, r_corrBatchSum, i_corrBatchValidCount, stream);

#ifdef CUAMPCOR_DEBUG
    r_maxval->outputToFile("r_maxval", stream);
    r_corrBatchRawZoomIn->outputToFile("r_corrBatchRawStatZoomIn", stream);
    i_corrBatchZoomInValid->outputToFile("i_corrBatchZoomInValid", stream);
    r_corrBatchSum->outputToFile("r_corrBatchSum", stream);
    i_corrBatchValidCount->outputToFile("i_corrBatchValidCount", stream);
#endif

    // step3: divide the peak value by the mean of surrounding values
    cuEstimateSnr(r_corrBatchSum, i_corrBatchValidCount, r_maxval, r_snrValue, stream);

#ifdef CUAMPCOR_DEBUG
    offsetInit->outputToFile("i_offsetInit", stream);
    r_snrValue->outputToFile("r_snrValue", stream);
    r_covValue->outputToFile("r_covValue", stream);
#endif

    // Using the approximate estimation to adjust secondary image (half search window size becomes only 4 pixels)
    // determine the starting pixel to extract secondary images around the max location
    cuDetermineSecondaryExtractOffset(offsetInit,
        maxLocShift,
        param->halfSearchRangeDownRaw, // old range
        param->halfSearchRangeAcrossRaw,
        param->halfZoomWindowSizeRaw,  // new range
        param->halfZoomWindowSizeRaw,
        stream);

#ifdef CUAMPCOR_DEBUG
    offsetInit->outputToFile("i_offsetInitAdjusted", stream);
    maxLocShift->outputToFile("i_maxLocShift", stream);
#endif

    // oversample reference
    // (deramping included in oversampler, but not necessary)
    referenceBatchOverSampler->execute(c_referenceBatchRaw, c_referenceBatchOverSampled, 0);

#ifdef CUAMPCOR_DEBUG
    // dump the oversampled reference image(s)
    c_referenceBatchOverSampled->outputToFile("c_referenceBatchOverSampled", stream);
#endif

    // compute and subtract the mean value
    cuArraysSubtractMean(c_referenceBatchOverSampled, stream);

#ifdef CUAMPCOR_DEBUG
    // dump the oversampled reference image(s) with mean subtracted
    c_referenceBatchOverSampled->outputToFile("c_referenceBatchOverSampledSubMean",stream);
#endif

    if(param->useHannWindowReference) {

        windowFilterReferenceOverSampled->filter(c_referenceBatchOverSampled, stream);
#ifdef CUAMPCOR_DEBUG
        // dump the filtered oversampled reference image(s) with mean subtracted
        c_referenceBatchOverSampled->outputToFile("c_referenceBatchOverSampledSubMeanFiltered",stream);
#endif
    }

    // extract secondary and oversample
    cuArraysCopyExtract(c_secondaryBatchRaw, c_secondaryBatchZoomIn, offsetInit, stream);
    secondaryBatchOverSampler->execute(c_secondaryBatchZoomIn, c_secondaryBatchOverSampled, 0);

#ifdef CUAMPCOR_DEBUG
    // dump the extracted raw secondary image
    c_secondaryBatchZoomIn->outputToFile("c_secondaryBatchZoomIn", stream);
    // dump the oversampled secondary image(s)
    c_secondaryBatchOverSampled->outputToFile("c_secondaryBatchOverSampled", stream);
#endif

    if(param->useHannWindowSecondary) {

        windowFilterSecondaryOverSampled->filter(c_secondaryBatchOverSampled, stream);
#ifdef CUAMPCOR_DEBUG
        // dump the filtered oversampled secondary image(s)
        c_secondaryBatchOverSampled->outputToFile("c_secondaryBatchOverSampledFiltered", stream);
#endif
    }

    // correlate oversampled images
    cuCorrFreqDomain_OverSampled->execute(c_referenceBatchOverSampled, c_secondaryBatchOverSampled, r_corrBatchZoomIn);

#ifdef CUAMPCOR_DEBUG
    // dump the oversampled correlation surface (un-normalized)
    r_corrBatchZoomIn->outputToFile("r_corrBatchZoomInUnNorm", stream);
#endif

    // normalize the correlation surface
    corrNormalizerOverSampled->execute(r_corrBatchZoomIn, c_referenceBatchOverSampled, c_secondaryBatchOverSampled, stream);

#ifdef CUAMPCOR_DEBUG
    // dump the oversampled correlation surface (normalized)
    r_corrBatchZoomIn->outputToFile("r_corrBatchZoomIn", stream);
#endif

    // remove the last row and col to get even sequences
    cuArraysCopyExtract(r_corrBatchZoomIn, r_corrBatchZoomInAdjust, make_int2(0,0), stream);

#ifdef CUAMPCOR_DEBUG
    // dump the adjusted correlation Surface
    r_corrBatchZoomInAdjust->outputToFile("r_corrBatchZoomInAdjust", stream);
#endif

    // oversample the correlation surface
    if(param->oversamplingMethod) {
        // sinc interpolator only computes (-i_sincwindow, i_sincwindow)*oversamplingfactor
        // we need the max loc as the center if shifted
        corrSincOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled,
            maxLocShift, param->oversamplingFactor*param->rawDataOversamplingFactor
            );
    }
    else {
        corrOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled);
    }

#ifdef CUAMPCOR_DEBUG
    // dump the oversampled correlation surface
    r_corrBatchZoomInOverSampled->outputToFile("r_corrBatchZoomInOverSampled", stream);
#endif

    //find the max again
    cuArraysMaxloc2D(r_corrBatchZoomInOverSampled, offsetZoomIn, corrMaxValue, stream);

#ifdef CUAMPCOR_DEBUG
    // dump the max location on oversampled correlation surface
    offsetZoomIn->outputToFile("i_offsetZoomIn", stream);
    corrMaxValue->outputToFile("r_maxvalZoomInOversampled", stream);
#endif

    // determine the final offset from non-oversampled (pixel) and oversampled (sub-pixel)
    // = (Init-HalfsearchRange) + ZoomIn/(2*ovs)
    cuSubPixelOffset(offsetInit, offsetZoomIn, offsetFinal,
        param->oversamplingFactor, param->rawDataOversamplingFactor,
        param->halfSearchRangeDownRaw, param->halfSearchRangeAcrossRaw,
        stream);

    // Insert the chunk results to final images
    cuArraysCopyInsert(offsetFinal, offsetImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
    // snr
    cuArraysCopyInsert(r_snrValue, snrImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
    // Variance.
    cuArraysCopyInsert(r_covValue, covImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
    // all done

}

/// set chunk index
void cuAmpcorChunk::setIndex(int idxDown_, int idxAcross_)
{
    idxChunkDown = idxDown_;
    idxChunkAcross = idxAcross_;
    idxChunk = idxChunkAcross + idxChunkDown*param->numberChunkAcross;

    if(idxChunkDown == param->numberChunkDown -1) {
        nWindowsDown = param->numberWindowDown - param->numberWindowDownInChunk*(param->numberChunkDown -1);
    }
    else {
        nWindowsDown = param->numberWindowDownInChunk;
    }

    if(idxChunkAcross == param->numberChunkAcross -1) {
        nWindowsAcross = param->numberWindowAcross - param->numberWindowAcrossInChunk*(param->numberChunkAcross -1);
    }
    else {
        nWindowsAcross = param->numberWindowAcrossInChunk;
    }
}

/// obtain the starting pixels for each chip
/// @param[in] oStartPixel start pixel locations for all chips
/// @param[out] rstartPixel  start pixel locations for chips within the chunk
void cuAmpcorChunk::getRelativeOffset(int *rStartPixel, const int *oStartPixel, int diff)
{
    for(int i=0; i<param->numberWindowDownInChunk; ++i) {
        int iDown = i;
        if(i>=nWindowsDown) iDown = nWindowsDown-1;
        for(int j=0; j<param->numberWindowAcrossInChunk; ++j){
            int iAcross = j;
            if(j>=nWindowsAcross) iAcross = nWindowsAcross-1;
            int idxInChunk = iDown*param->numberWindowAcrossInChunk+iAcross;
            int idxInAll = (iDown+idxChunkDown*param->numberWindowDownInChunk)*param->numberWindowAcross
                + idxChunkAcross*param->numberWindowAcrossInChunk+iAcross;
            rStartPixel[idxInChunk] = oStartPixel[idxInAll] - diff;
        }
    }
}

void cuAmpcorChunk::loadReferenceChunk()
{

    // we first load the whole chunk of image from cpu to a gpu buffer c(r)_referenceChunkRaw
    // then copy to a batch of windows with (nImages, height, width) (leading dimension on the right)

    // get the chunk size to be loaded to gpu
    int startD = param->referenceChunkStartPixelDown[idxChunk]; //start pixel down (along height)
    int startA = param->referenceChunkStartPixelAcross[idxChunk]; // start pixel across (along width)
    int height =  param->referenceChunkHeight[idxChunk]; // number of pixels along height
    int width = param->referenceChunkWidth[idxChunk];  // number of pixels along width

    //use cpu to compute the starting positions for each window
    getRelativeOffset(ChunkOffsetDown->hostData, param->referenceStartPixelDown, param->referenceChunkStartPixelDown[idxChunk]);
    // copy the positions to gpu
    ChunkOffsetDown->copyToDevice(stream);
    // same for the across direction
    getRelativeOffset(ChunkOffsetAcross->hostData, param->referenceStartPixelAcross, param->referenceChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);

    // check whether the image is complex (e.g., SLC) or real( e.g. TIFF)
    if(referenceImage->isComplex())
    {
        // allocate a gpu buffer to load data from cpu/file
        // try allocate/deallocate the buffer on the fly to save gpu memory 07/09/19
        c_referenceChunkRaw = new cuArrays<float2> (param->maxReferenceChunkHeight, param->maxReferenceChunkWidth);
        c_referenceChunkRaw->allocate();

        // load the data from cpu
        referenceImage->loadToDevice((void *)c_referenceChunkRaw->devData, startD, startA, height, width, stream);

        //copy the chunk to a batch format (nImages, height, width)
        // if derampMethod = 0 (no deramp), take amplitudes; otherwise, copy complex data

        cuArraysCopyToBatchWithOffset(c_referenceChunkRaw, param->referenceChunkWidth[idxChunk],
            c_referenceBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        // deallocate the gpu buffer
        c_referenceChunkRaw->deallocate();
    }
    else {
        std::cout << "Error: the image is not complex\n";
    }

}

void cuAmpcorChunk::loadSecondaryChunk()
{

    // get the reading offset (starting pixels for each image)
    getRelativeOffset(ChunkOffsetDown->hostData, param->secondaryStartPixelDown, param->secondaryChunkStartPixelDown[idxChunk]);
    ChunkOffsetDown->copyToDevice(stream);
    getRelativeOffset(ChunkOffsetAcross->hostData, param->secondaryStartPixelAcross, param->secondaryChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);

    if(secondaryImage->isComplex())
    {
        // allocate reading buffer on-the-fly
        c_secondaryChunkRaw = new cuArrays<float2> (param->maxSecondaryChunkHeight, param->maxSecondaryChunkWidth);
        c_secondaryChunkRaw->allocate();

        //load a chunk from mmap to gpu
        secondaryImage->loadToDevice(c_secondaryChunkRaw->devData,
            param->secondaryChunkStartPixelDown[idxChunk],
            param->secondaryChunkStartPixelAcross[idxChunk],
            param->secondaryChunkHeight[idxChunk],
            param->secondaryChunkWidth[idxChunk],
            stream);
        // copy images into batched format
        cuArraysCopyToBatchWithOffset(c_secondaryChunkRaw, param->secondaryChunkWidth[idxChunk],
            c_secondaryBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        // deallocate buffer
        c_secondaryChunkRaw->deallocate();
    }
    else {
        std::cout << "Error: the image is not complex\n";
    }
}

/// constructor
cuAmpcorChunk::cuAmpcorChunk(cuAmpcorParameter *param_, GDALImage *reference_, GDALImage *secondary_,
    cuArrays<float2> *offsetImage_, cuArrays<float> *snrImage_, cuArrays<float3> *covImage_,
    hipStream_t stream_)

{
    param = param_;
    referenceImage = reference_;
    secondaryImage = secondary_;
    offsetImage = offsetImage_;
    snrImage = snrImage_;
    covImage = covImage_;

    stream = stream_;

    ChunkOffsetDown = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetDown->allocate();
    ChunkOffsetDown->allocateHost();
    ChunkOffsetAcross = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetAcross->allocate();
    ChunkOffsetAcross->allocateHost();

    c_referenceChunkRaw = new cuArrays<float2> (param->maxReferenceChunkHeight, param->maxReferenceChunkWidth);
    c_referenceChunkRaw->allocate();

    c_referenceBatchRaw = new cuArrays<float2> (
        param->windowSizeHeightRaw, param->windowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_referenceBatchRaw->allocate();

    c_secondaryBatchRaw = new cuArrays<float2> (
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_secondaryBatchRaw->allocate();

    c_secondaryBatchZoomIn = new cuArrays<float2> (
        param->searchWindowSizeHeightRawZoomIn, param->searchWindowSizeWidthRawZoomIn,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_secondaryBatchZoomIn->allocate();

    c_referenceBatchOverSampled = new cuArrays<float2> (
            param->windowSizeHeight, param->windowSizeWidth,
            param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_referenceBatchOverSampled->allocate();

    c_secondaryBatchOverSampled = new cuArrays<float2> (
            param->searchWindowSizeHeight, param->searchWindowSizeWidth,
            param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_secondaryBatchOverSampled->allocate();


    referenceBatchOverSampler = new cuOverSamplerC2C(
        c_referenceBatchRaw->height, c_referenceBatchRaw->width, //original size
        c_referenceBatchOverSampled->height, c_referenceBatchOverSampled->width, //oversampled size
        c_referenceBatchRaw->count, stream);

    secondaryBatchOverSampler = new cuOverSamplerC2C(c_secondaryBatchZoomIn->height, c_secondaryBatchZoomIn->width,
            c_secondaryBatchOverSampled->height, c_secondaryBatchOverSampled->width, c_secondaryBatchRaw->count, stream);

    r_corrBatchRaw = new cuArrays<float> (
            param->searchWindowSizeHeightRaw-param->windowSizeHeightRaw+1,
            param->searchWindowSizeWidthRaw-param->windowSizeWidthRaw+1,
            param->numberWindowDownInChunk,
            param->numberWindowAcrossInChunk);
    r_corrBatchRaw->allocate();

    r_corrBatchZoomIn = new cuArrays<float> (
            param->searchWindowSizeHeight - param->windowSizeHeight+1,
            param->searchWindowSizeWidth - param->windowSizeWidth+1,
            param->numberWindowDownInChunk,
            param->numberWindowAcrossInChunk);
    r_corrBatchZoomIn->allocate();

    r_corrBatchZoomInAdjust = new cuArrays<float> (
            param->searchWindowSizeHeight - param->windowSizeHeight,
            param->searchWindowSizeWidth - param->windowSizeWidth,
            param->numberWindowDownInChunk,
            param->numberWindowAcrossInChunk);
    r_corrBatchZoomInAdjust->allocate();


    r_corrBatchZoomInOverSampled = new cuArrays<float> (
        param->zoomWindowSize * param->oversamplingFactor,
        param->zoomWindowSize * param->oversamplingFactor,
        param->numberWindowDownInChunk,
        param->numberWindowAcrossInChunk);
    r_corrBatchZoomInOverSampled->allocate();

    offsetInit = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetInit->allocate();

    offsetZoomIn = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetZoomIn->allocate();

    offsetFinal = new cuArrays<float2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetFinal->allocate();

    maxLocShift = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    maxLocShift->allocate();

    corrMaxValue = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    corrMaxValue->allocate();


    // new arrays due to snr estimation
    r_corrBatchRawZoomIn = new cuArrays<float> (
            param->corrRawZoomInHeight,
            param->corrRawZoomInWidth,
            param->numberWindowDownInChunk,
            param->numberWindowAcrossInChunk);
    r_corrBatchRawZoomIn->allocate();

    i_corrBatchZoomInValid = new cuArrays<int> (
            param->corrRawZoomInHeight,
            param->corrRawZoomInWidth,
            param->numberWindowDownInChunk,
            param->numberWindowAcrossInChunk);
    i_corrBatchZoomInValid->allocate();


    r_corrBatchSum = new cuArrays<float> (
                    param->numberWindowDownInChunk,
                    param->numberWindowAcrossInChunk);
    r_corrBatchSum->allocate();

    i_corrBatchValidCount = new cuArrays<int> (
                        param->numberWindowDownInChunk,
                        param->numberWindowAcrossInChunk);
    i_corrBatchValidCount->allocate();

    i_maxloc = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    i_maxloc->allocate();

    r_maxval = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_maxval->allocate();

    r_snrValue = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_snrValue->allocate();

    r_covValue = new cuArrays<float3> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_covValue->allocate();

    // end of new arrays

    if(param->oversamplingMethod) {
        corrSincOverSampler = new cuSincOverSamplerR2R(param->oversamplingFactor, stream);
    }
    else {
        corrOverSampler= new cuOverSamplerR2R(param->zoomWindowSize, param->zoomWindowSize,
            (param->zoomWindowSize)*param->oversamplingFactor,
            (param->zoomWindowSize)*param->oversamplingFactor,
            param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
            stream);
    }

    cuCorrFreqDomain = new cuFreqCorrelator(
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
        stream);
    cuCorrFreqDomain_OverSampled = new cuFreqCorrelator(
            param->searchWindowSizeHeight, param->searchWindowSizeWidth,
            param->numberWindowDownInChunk * param->numberWindowAcrossInChunk,
            stream);

    corrNormalizerRaw = new cuNormalizeSAT(
        param->searchWindowSizeHeightRaw,
        param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk * param->numberWindowAcrossInChunk
        );

    corrNormalizerOverSampled = new cuNormalizeSAT(
        param->searchWindowSizeHeight,
        param->searchWindowSizeWidth,
        param->numberWindowDownInChunk * param->numberWindowAcrossInChunk
        );

    // if Hann Filter on reference (both raw and oversampled) is desired
    if (param->useHannWindowReference) {
        // raw
        windowFilterReferenceRaw = new cuHannWindowFilter(
            param->windowSizeHeightRaw, param->windowSizeWidthRaw);
        c_referenceBatchRawFiltered = new cuArrays<float2>(
            param->windowSizeHeightRaw, param->windowSizeWidthRaw,
            param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
        c_referenceBatchRawFiltered -> allocate();
        // oversampled
        windowFilterReferenceOverSampled = new cuHannWindowFilter(
            param->windowSizeHeight, param->windowSizeWidth);
    }
    else {
        //if not, make a link of the data
        c_referenceBatchRawFiltered = c_referenceBatchRaw;
    }
    // same for the secondary image
    if (param->useHannWindowSecondary) {
        windowFilterSecondaryRaw = new cuHannWindowFilter(
            param->searchWindowSizeHeight, param->searchWindowSizeWidthRaw);
        c_secondaryBatchRawFiltered = new cuArrays<float2>(
            param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
            param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
        c_secondaryBatchRawFiltered->allocate();
        windowFilterSecondaryOverSampled = new cuHannWindowFilter(
            param->searchWindowSizeHeight, param->searchWindowSizeWidth);
    }
    else {
        c_secondaryBatchRawFiltered = c_secondaryBatchRaw;
    }


#ifdef CUAMPCOR_DEBUG
    std::cout << "all objects in chunk are created ...\n";
#endif
}

// destructor
cuAmpcorChunk::~cuAmpcorChunk()
{
}

// end of file
